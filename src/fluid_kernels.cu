#include "hip/hip_runtime.h"
//
// Created by quan on 4/16/17.
//

#include <fluid_kernels.h>
#include <>

__global__ void advect_kernel(double *d, double *d0, double *velocX, double *velocY,
                              double *velocZ, double dt, int N, int k)
{
    double Ndouble = (double) N;
    double dtx = dt * (N - 2);
    double dty = dt * (N - 2);
    double dtz = dt * (N - 2);

    int j = blockIdx.x + 1;
    int i = threadIdx.x + 1;
    double idouble = (double) i;
    double jdouble = (double) j;
    double kdouble = (double) k;

    double s0, s1, t0, t1, u0, u1;
    double tmp1, tmp2, tmp3, x, y, z;
    double i0, i1, j0, j1, k0, k1;

    tmp1 = dtx * velocX[IX(i, j, k)];
    tmp2 = dty * velocY[IX(i, j, k)];
    tmp3 = dtz * velocZ[IX(i, j, k)];
    x    = idouble - tmp1;
    y    = jdouble - tmp2;
    z    = kdouble - tmp3;

    if(x < 0.5f) x = 0.5f;
    if(x > Ndouble + 0.5f) x = Ndouble + 0.5f;
    i0 = floor(x);
    i1 = i0 + 1.0f;
    if(y < 0.5f) y = 0.5f;
    if(y > Ndouble + 0.5f) y = Ndouble + 0.5f;
    j0 = floor(y);
    j1 = j0 + 1.0f;
    if(z < 0.5f) z = 0.5f;
    if(z > Ndouble + 0.5f) z = Ndouble + 0.5f;
    k0 = floor(z);
    k1 = k0 + 1.0f;

    s1 = x - i0;
    s0 = 1.0f - s1;
    t1 = y - j0;
    t0 = 1.0f - t1;
    u1 = z - k0;
    u0 = 1.0f - u1;

    int i0i = (int) i0;
    int i1i = (int) i1;
    int j0i = (int) j0;
    int j1i = (int) j1;
    int k0i = (int) k0;
    int k1i = (int) k1;

    d[IX(i, j, k)] =

            s0 * ( t0 * (u0 * d0[IX(i0i, j0i, k0i)]
                         +u1 * d0[IX(i0i, j0i, k1i)])
                   +( t1 * (u0 * d0[IX(i0i, j1i, k0i)]
                            +u1 * d0[IX(i0i, j1i, k1i)])))
            +s1 * ( t0 * (u0 * d0[IX(i1i, j0i, k0i)]
                          +u1 * d0[IX(i1i, j0i, k1i)])
                    +( t1 * (u0 * d0[IX(i1i, j1i, k0i)]
                             +u1 * d0[IX(i1i, j1i, k1i)])));
}

__global__ void set_bnd_kernel1(int b, double *x, int N)
{
    int j = blockIdx.x + 1;
    int i = threadIdx.x + 1;

    x[IX(i, j, 0  )] = b == 3 ? -x[IX(i, j, 1  )] : x[IX(i, j, 1  )];
    x[IX(i, j, N-1)] = b == 3 ? -x[IX(i, j, N-2)] : x[IX(i, j, N-2)];

    x[IX(i, 0  , j)] = b == 2 ? -x[IX(i, 1  , j)] : x[IX(i, 1  , j)];
    x[IX(i, N-1, j)] = b == 2 ? -x[IX(i, N-2, j)] : x[IX(i, N-2, j)];

    x[IX(0  , i, j)] = b == 1 ? -x[IX(1  , i, j)] : x[IX(1  , i, j)];
    x[IX(N-1, i, j)] = b == 1 ? -x[IX(N-2, i, j)] : x[IX(N-2, i, j)];
}

__global__ void set_bnd_kernel2(int b, double *x, int N)
{
    int k = blockIdx.x + 1;
    int i = threadIdx.x + 1;

    x[IX(i, 0  , k)] = b == 2 ? -x[IX(i, 1  , k)] : x[IX(i, 1  , k)];
    x[IX(i, N-1, k)] = b == 2 ? -x[IX(i, N-2, k)] : x[IX(i, N-2, k)];
}

__global__ void set_bnd_kernel3(int b, double *x, int N)
{
    int j = blockIdx.x + 1;
    int k = threadIdx.x + 1;

    x[IX(0  , j, k)] = b == 1 ? -x[IX(1  , j, k)] : x[IX(1  , j, k)];
    x[IX(N-1, j, k)] = b == 1 ? -x[IX(N-2, j, k)] : x[IX(N-2, j, k)];
}
