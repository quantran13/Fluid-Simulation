#include "hip/hip_runtime.h"
#include <fluid.h>

#define SERIAL_SET_BND 0
#define SERIAL_ADVECT 0
#define SERIAL_PROJECT 0
#define SERIAL_LIN_SOLVE 0

FluidCube *FluidCubeCreate(int size, int diffusion, int viscosity, double dt)
{
    FluidCube *cube = (FluidCube *) malloc(sizeof(*cube));
    size_t N = (size_t) size;

    cube->size = size;
    cube->dt = dt;
    cube->diff = diffusion;
    cube->visc = viscosity;

    hipMallocManaged((void **) &cube->s, N * N * N * sizeof(double));
    hipMallocManaged((void **) &cube->density, N * N * N * sizeof(double));

    hipMallocManaged((void **) &cube->Vx, N * N * N * sizeof(double));
    hipMallocManaged((void **) &cube->Vy, N * N * N * sizeof(double));
    hipMallocManaged((void **) &cube->Vz, N * N * N * sizeof(double));

    hipMallocManaged((void **) &cube->Vx0, N * N * N * sizeof(double));
    hipMallocManaged((void **) &cube->Vy0, N * N * N * sizeof(double));
    hipMallocManaged((void **) &cube->Vz0, N * N * N * sizeof(double));

    return cube;
}

void FluidCubeFree(FluidCube *cube)
{
    hipFree(cube->s);
    hipFree(cube->density);

    hipFree(cube->Vx);
    hipFree(cube->Vy);
    hipFree(cube->Vz);

    hipFree(cube->Vx0);
    hipFree(cube->Vy0);
    hipFree(cube->Vz0);

    free(cube);
}

static void set_bnd_serial(int b, double *x, int N)
{
    for(int j = 1; j < N - 1; j++) {
        for(int i = 1; i < N - 1; i++) {
            x[IX(i, j, 0  )] = b == 3 ? -x[IX(i, j, 1  )] : x[IX(i, j, 1  )];
            x[IX(i, j, N-1)] = b == 3 ? -x[IX(i, j, N-2)] : x[IX(i, j, N-2)];
        }
    }
    for(int k = 1; k < N - 1; k++) {
        for(int i = 1; i < N - 1; i++) {
            x[IX(i, 0  , k)] = b == 2 ? -x[IX(i, 1  , k)] : x[IX(i, 1  , k)];
            x[IX(i, N-1, k)] = b == 2 ? -x[IX(i, N-2, k)] : x[IX(i, N-2, k)];
        }
    }
    for(int k = 1; k < N - 1; k++) {
        for(int j = 1; j < N - 1; j++) {
            x[IX(0  , j, k)] = b == 1 ? -x[IX(1  , j, k)] : x[IX(1  , j, k)];
            x[IX(N-1, j, k)] = b == 1 ? -x[IX(N-2, j, k)] : x[IX(N-2, j, k)];
        }
    }

    x[IX(0, 0, 0)]       = 0.33f * (x[IX(1, 0, 0)]
                                    + x[IX(0, 1, 0)]
                                    + x[IX(0, 0, 1)]);
    x[IX(0, N-1, 0)]     = 0.33f * (x[IX(1, N-1, 0)]
                                    + x[IX(0, N-2, 0)]
                                    + x[IX(0, N-1, 1)]);
    x[IX(0, 0, N-1)]     = 0.33f * (x[IX(1, 0, N-1)]
                                    + x[IX(0, 1, N-1)]
                                    + x[IX(0, 0, N)]);
    x[IX(0, N-1, N-1)]   = 0.33f * (x[IX(1, N-1, N-1)]
                                    + x[IX(0, N-2, N-1)]
                                    + x[IX(0, N-1, N-2)]);
    x[IX(N-1, 0, 0)]     = 0.33f * (x[IX(N-2, 0, 0)]
                                    + x[IX(N-1, 1, 0)]
                                    + x[IX(N-1, 0, 1)]);
    x[IX(N-1, N-1, 0)]   = 0.33f * (x[IX(N-2, N-1, 0)]
                                    + x[IX(N-1, N-2, 0)]
                                    + x[IX(N-1, N-1, 1)]);
    x[IX(N-1, 0, N-1)]   = 0.33f * (x[IX(N-2, 0, N-1)]
                                    + x[IX(N-1, 1, N-1)]
                                    + x[IX(N-1, 0, N-2)]);
    x[IX(N-1, N-1, N-1)] = 0.33f * (x[IX(N-2, N-1, N-1)]
                                    + x[IX(N-1, N-2, N-1)]
                                    + x[IX(N-1, N-1, N-2)]);
}

static void set_bnd(int b, double *x, int N)
{
#if SERIAL_SET_BND
    for(int j = 1; j < N - 1; j++) {
        for(int i = 1; i < N - 1; i++) {
            x[IX(i, j, 0  )] = b == 3 ? -x[IX(i, j, 1  )] : x[IX(i, j, 1  )];
            x[IX(i, j, N-1)] = b == 3 ? -x[IX(i, j, N-2)] : x[IX(i, j, N-2)];
        }
    }
    for(int k = 1; k < N - 1; k++) {
        for(int i = 1; i < N - 1; i++) {
            x[IX(i, 0  , k)] = b == 2 ? -x[IX(i, 1  , k)] : x[IX(i, 1  , k)];
            x[IX(i, N-1, k)] = b == 2 ? -x[IX(i, N-2, k)] : x[IX(i, N-2, k)];
        }
    }
    for(int k = 1; k < N - 1; k++) {
        for(int j = 1; j < N - 1; j++) {
            x[IX(0  , j, k)] = b == 1 ? -x[IX(1  , j, k)] : x[IX(1  , j, k)];
            x[IX(N-1, j, k)] = b == 1 ? -x[IX(N-2, j, k)] : x[IX(N-2, j, k)];
        }
    }

    x[IX(0, 0, 0)]       = 0.33f * (x[IX(1, 0, 0)]
                                  + x[IX(0, 1, 0)]
                                  + x[IX(0, 0, 1)]);
    x[IX(0, N-1, 0)]     = 0.33f * (x[IX(1, N-1, 0)]
                                  + x[IX(0, N-2, 0)]
                                  + x[IX(0, N-1, 1)]);
    x[IX(0, 0, N-1)]     = 0.33f * (x[IX(1, 0, N-1)]
                                  + x[IX(0, 1, N-1)]
                                  + x[IX(0, 0, N)]);
    x[IX(0, N-1, N-1)]   = 0.33f * (x[IX(1, N-1, N-1)]
                                  + x[IX(0, N-2, N-1)]
                                  + x[IX(0, N-1, N-2)]);
    x[IX(N-1, 0, 0)]     = 0.33f * (x[IX(N-2, 0, 0)]
                                  + x[IX(N-1, 1, 0)]
                                  + x[IX(N-1, 0, 1)]);
    x[IX(N-1, N-1, 0)]   = 0.33f * (x[IX(N-2, N-1, 0)]
                                  + x[IX(N-1, N-2, 0)]
                                  + x[IX(N-1, N-1, 1)]);
    x[IX(N-1, 0, N-1)]   = 0.33f * (x[IX(N-2, 0, N-1)]
                                  + x[IX(N-1, 1, N-1)]
                                  + x[IX(N-1, 0, N-2)]);
    x[IX(N-1, N-1, N-1)] = 0.33f * (x[IX(N-2, N-1, N-1)]
                                  + x[IX(N-1, N-2, N-1)]
                                  + x[IX(N-1, N-1, N-2)]);
#endif

#if not SERIAL_SET_BND
    set_bnd_kernel1 <<< N-2, N-2 >>> (b, x, N);
    set_bnd_kernel2 <<< 1, 1 >>> (x, N);
    hipDeviceSynchronize();
#endif
}

static void lin_solve(int b, double *x, double *x0, double a, double c, int N)
{
    double cRecip = 1.0 / c;
    int iter = 4;

#if not SERIAL_LIN_SOLVE
    double *x_next;
    hipMallocManaged((void **) &x_next, N * N * N * sizeof(double));

    for (int k = 0; k < iter; k++) {
        for (int m = 1; m < N - 1; m++) {
            lin_solve_kernel <<< N-2, N-2 >>> (x_next, x, x0, a, cRecip, N, m);
        }

        for (int m = 1; m < N - 1; m++) {
            set_values_kernel <<< N-2, N-2 >>> (x_next, x, m, N);
        }

        set_bnd(b, x, N);
    }

    hipFree(x_next);
#endif

#if SERIAL_LIN_SOLVE
    for (int k = 0; k < iter; k++) {
        for (int m = 1; m < N - 1; m++) {
            for (int j = 1; j < N - 1; j++) {
                for (int i = 1; i < N - 1; i++) {
                    x[IX(i, j, m)] =
                            (x0[IX(i, j, m)]
                             + a * (x[IX(i + 1, j, m)]
                                    + x[IX(i - 1, j, m)]
                                    + x[IX(i, j + 1, m)]
                                    + x[IX(i, j - 1, m)]
                                    + x[IX(i, j, m + 1)]
                                    + x[IX(i, j, m - 1)]
                            )) * cRecip;
                }
            }
        }

        set_bnd_serial(b, x, N);
        //set_bnd(b, x, N);
    }
#endif
}

static void diffuse(int b, double *x, double *x0, double diff, double dt, int N)
{
    double a = dt * diff * (N - 2) * (N - 2);
    lin_solve(b, x, x0, a, 1 + 6 * a, N);
}

static void advect(int b, double *d, double *d0, double *velocX,
                   double *velocY, double *velocZ, double dt, int N)
{
#if SERIAL_ADVECT
    double i0, i1, j0, j1, k0, k1;

    double dtx = dt * (N - 2);
    double dty = dt * (N - 2);
    double dtz = dt * (N - 2);

    double s0, s1, t0, t1, u0, u1;
    double tmp1, tmp2, tmp3, x, y, z;

    double Ndouble = N;
    double idouble, jdouble, kdouble;
    int i, j, k;

    for(k = 1, kdouble = 1; k < N - 1; k++, kdouble++) {
        for(j = 1, jdouble = 1; j < N - 1; j++, jdouble++) {
            for(i = 1, idouble = 1; i < N - 1; i++, idouble++) {
                tmp1 = dtx * velocX[IX(i, j, k)];
                tmp2 = dty * velocY[IX(i, j, k)];
                tmp3 = dtz * velocZ[IX(i, j, k)];
                x    = idouble - tmp1;
                y    = jdouble - tmp2;
                z    = kdouble - tmp3;

                if(x < 0.5f) x = 0.5f;
                if(x > Ndouble + 0.5f) x = Ndouble + 0.5f;
                i0 = floorf(x);
                i1 = i0 + 1.0f;
                if(y < 0.5f) y = 0.5f;
                if(y > Ndouble + 0.5f) y = Ndouble + 0.5f;
                j0 = floorf(y);
                j1 = j0 + 1.0f;
                if(z < 0.5f) z = 0.5f;
                if(z > Ndouble + 0.5f) z = Ndouble + 0.5f;
                k0 = floorf(z);
                k1 = k0 + 1.0f;

                s1 = x - i0;
                s0 = 1.0f - s1;
                t1 = y - j0;
                t0 = 1.0f - t1;
                u1 = z - k0;
                u0 = 1.0f - u1;

                int i0i = (int) i0;
                int i1i = (int) i1;
                int j0i = (int) j0;
                int j1i = (int) j1;
                int k0i = (int) k0;
                int k1i = (int) k1;

                d[IX(i, j, k)] =

                        s0 * ( t0 * (u0 * d0[IX(i0i, j0i, k0i)]
                                     +u1 * d0[IX(i0i, j0i, k1i)])
                               +( t1 * (u0 * d0[IX(i0i, j1i, k0i)]
                                        +u1 * d0[IX(i0i, j1i, k1i)])))
                        +s1 * ( t0 * (u0 * d0[IX(i1i, j0i, k0i)]
                                      +u1 * d0[IX(i1i, j0i, k1i)])
                                +( t1 * (u0 * d0[IX(i1i, j1i, k0i)]
                                         +u1 * d0[IX(i1i, j1i, k1i)])));
            }
        }
    }
#endif

#if not SERIAL_ADVECT
    for (int k = 1; k < N - 1; k++) {
        advect_kernel <<< N-2, N-2 >>> (d, d0, velocX, velocY, velocZ, dt, N, k);
    }
    hipDeviceSynchronize();
#endif

    set_bnd(b, d, N);
}

static void project(double *velocX, double *velocY, double *velocZ,
                    double *p, double *div, int N)
{
    double N_recip = 1 / N;
    for (int k = 1; k < N - 1; k++) {
#if not SERIAL_PROJECT
        project_kernel1 <<< N-2, N-2 >>> (velocX, velocY, velocZ, p, div,
                                          N, N_recip, k);
#endif

#if SERIAL_PROJECT
        for (int j = 1; j < N - 1; j++) {
            for (int i = 1; i < N - 1; i++) {
                div[IX(i, j, k)] = -0.5f*(
                        velocX[IX(i+1, j  , k  )]
                       -velocX[IX(i-1, j  , k  )]
                       +velocY[IX(i  , j+1, k  )]
                       -velocY[IX(i  , j-1, k  )]
                       +velocZ[IX(i  , j  , k+1)]
                       -velocZ[IX(i  , j  , k-1)]
                   ) * N_recip;
                p[IX(i, j, k)] = 0;
            }
        }
#endif
    }

#if not SERIAL_PROJECT
    hipDeviceSynchronize();
#endif

    set_bnd(0, div, N);
    set_bnd(0, p, N);
    lin_solve(0, p, div, 1, 6, N);

    for (int k = 1; k < N - 1; k++) {
#if SERIAL_PROJECT
        for (int j = 1; j < N - 1; j++) {
            for (int i = 1; i < N - 1; i++) {
                velocX[IX(i, j, k)] -= 0.5f * (p[IX(i+1, j, k)]
                                               - p[IX(i-1, j, k)]) * N;
                velocY[IX(i, j, k)] -= 0.5f * (p[IX(i, j+1, k)]
                                               - p[IX(i, j-1, k)]) * N;
                velocZ[IX(i, j, k)] -= 0.5f * (p[IX(i, j, k+1)]
                                               - p[IX(i, j, k-1)]) * N;
            }
        }
#endif

#if not SERIAL_PROJECT
        project_kernel2 <<< N-2, N-2 >>> (velocX, velocY, velocZ, p, N, k);
#endif
    }

#if not SERIAL_PROJECT
    hipDeviceSynchronize();
#endif

    set_bnd(1, velocX, N);
    set_bnd(2, velocY, N);
    set_bnd(3, velocZ, N);
}

void FluidCubeStep(FluidCube *cube, perf_t *perf_struct)
{
    int N = cube->size;
    double visc = cube->visc;
    double diff = cube->diff;
    double dt = cube->dt;
    double *Vx = cube->Vx;
    double *Vy = cube->Vy;
    double *Vz = cube->Vz;
    double *Vx0 = cube->Vx0;
    double *Vy0 = cube->Vy0;
    double *Vz0 = cube->Vz0;
    double *s = cube->s;
    double *density = cube->density;

    double start = 0, end = 0;

    start = get_time();
    diffuse(1, Vx0, Vx, visc, dt, N);
    end = get_time();
    perf_struct->timeDiffuse += end - start;

    start = get_time();
    diffuse(2, Vy0, Vy, visc, dt, N);
    end = get_time();
    perf_struct->timeDiffuse += end - start;

    start = get_time();
    diffuse(3, Vz0, Vz, visc, dt, N);
    end = get_time();
    perf_struct->timeDiffuse += end - start;

    start = get_time();
    project(Vx0, Vy0, Vz0, Vx, Vy, N);
    end = get_time();
    perf_struct->timeProject += end - start;

    start = get_time();
    advect(1, Vx, Vx0, Vx0, Vy0, Vz0, dt, N);
    end = get_time();
    perf_struct->timeAdvect += end - start;

    start = get_time();
    advect(2, Vy, Vy0, Vx0, Vy0, Vz0, dt, N);
    end = get_time();
    perf_struct->timeAdvect += end - start;

    start = get_time();
    advect(3, Vz, Vz0, Vx0, Vy0, Vz0, dt, N);
    end = get_time();
    perf_struct->timeAdvect += end - start;

    start = get_time();
    project(Vx, Vy, Vz, Vx0, Vy0, N);
    end = get_time();
    perf_struct->timeProject += end - start;

    start = get_time();
    diffuse(0, s, density, diff, dt, N);
    end = get_time();
    perf_struct->timeDiffuse += end - start;

    start = get_time();
    advect(0, density, s, Vx, Vy, Vz, dt, N);
    end = get_time();
    perf_struct->timeAdvect += end - start;

    perf_struct->totalDiffuse += 4;
    perf_struct->totalAdvect += 4;
    perf_struct->totalProject += 2;
}

void FluidCubeAddDensity(FluidCube *cube, int x, int y, int z, double amount)
{
    int N = cube->size;
    cube->density[IX(x, y, z)] += amount;
}

void FluidCubeAddVelocity(FluidCube *cube, int x, int y, int z,
                          double amountX, double amountY, double amountZ)
{
    int N = cube->size;
    int index = IX(x, y, z);

    cube->Vx[index] += amountX;
    cube->Vy[index] += amountY;
    cube->Vz[index] += amountZ;
}
